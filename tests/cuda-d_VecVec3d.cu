#include "hip/hip_runtime.h"
/**
 * Demonstrates how to use the d_VecVec3d class on CUDA devices.
 */ 

#include <iostream>
#include <vector>
#include <string>


#include "types.hpp"
#include "types.cuh"
using namespace types;


__global__ void increment(d_VecVec3d d_vec){
    for(size_t i=0; i<d_vec.size(); ++i){
        d_vec(i,0) += 1;
    }
}

int main(){

    VecVec3d v = VecVec3d(10,1.0);
    std::cout << "Before increment: " << v(0,0) << std::endl;

    // Set up device vector using v
    d_VecVec3d d_vec = d_VecVec3d(&v);

    // increment vector on device
    increment<<<1,1>>>(d_vec);

    // copy back to host
    v.copy(d_vec.host_vec());
    std::cout << "After increment: " << v(0,0) << std::endl;

    return 0;
}


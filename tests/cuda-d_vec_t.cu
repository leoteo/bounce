#include "hip/hip_runtime.h"
/**
 * Demonstrates how to use the d_vec_t class on CUDA devices.
 */
#include <iostream>


#include "types.hpp"
#include "types.cuh"
using namespace types;

template<class T>
__global__ void increment(d_vec_t<T> d_vec){
    for(size_t i=0; i<d_vec.size(); ++i){
        d_vec(i) += 1;
    }
}

int main(){

    vec_t<real_t> v = vec_t<real_t>(10,1.0);
    std::cout << "Before increment: " << v(9) << std::endl;

    // Set up device vector using v
    d_vec_t<real_t> d_vec = d_vec_t<real_t>(&v);

    // increment vector on device
    increment<<<1,1>>>(d_vec);
    increment<<<1,1>>>(d_vec);
 
    // Old way, required members of d_vec_t to be public
    //vec_t<real_t> w(&d_vec);
    //std::cout << "After increment: " << w(0) << std::endl;

    //vec_t<real_t> x(d_vec.host_vec());
    //std::cout << "After increment: " << x(9) << std::endl;
    v.copy(d_vec.host_vec());
    std::cout << "After increment: " << v(9) << std::endl;


    return 0;
}

